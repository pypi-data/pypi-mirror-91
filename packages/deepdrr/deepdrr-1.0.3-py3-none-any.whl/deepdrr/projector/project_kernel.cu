#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cubicTex3D.cu>

#ifndef NUM_MATERIALS
#define NUM_MATERIALS 14
#endif

#define _seg(n) seg_##n
#define seg(n) _seg(n)

// channel of the materials array, same size as the volume.
#if NUM_MATERIALS > 0
texture<float, 3, hipReadModeElementType> seg(0);
#endif
#if NUM_MATERIALS > 1
texture<float, 3, hipReadModeElementType> seg(1);
#endif
#if NUM_MATERIALS > 2
texture<float, 3, hipReadModeElementType> seg(2);
#endif
#if NUM_MATERIALS > 3
texture<float, 3, hipReadModeElementType> seg(3);
#endif
#if NUM_MATERIALS > 4
texture<float, 3, hipReadModeElementType> seg(4);
#endif
#if NUM_MATERIALS > 5
texture<float, 3, hipReadModeElementType> seg(5);
#endif
#if NUM_MATERIALS > 6
texture<float, 3, hipReadModeElementType> seg(6);
#endif
#if NUM_MATERIALS > 7
texture<float, 3, hipReadModeElementType> seg(7);
#endif
#if NUM_MATERIALS > 8
texture<float, 3, hipReadModeElementType> seg(8);
#endif
#if NUM_MATERIALS > 9
texture<float, 3, hipReadModeElementType> seg(9);
#endif
#if NUM_MATERIALS > 10
texture<float, 3, hipReadModeElementType> seg(10);
#endif
#if NUM_MATERIALS > 11
texture<float, 3, hipReadModeElementType> seg(11);
#endif
#if NUM_MATERIALS > 12
texture<float, 3, hipReadModeElementType> seg(12);
#endif
#if NUM_MATERIALS > 13
texture<float, 3, hipReadModeElementType> seg(13);
#endif

#define UPDATE(multiplier, n) ({\
    output[idx + (n)] += (multiplier) * tex3D(volume, px, py, pz) * round(cubicTex3D(seg(n), px, py, pz));\
})

#if NUM_MATERIALS == 1
#define INTERPOLATE(multiplier) ({\
    UPDATE(multiplier, 0);\
})
#elif NUM_MATERIALS == 2
#define INTERPOLATE(multiplier) ({\
    UPDATE(multiplier, 0);\
    UPDATE(multiplier, 1);\
})
#elif NUM_MATERIALS == 3
#define INTERPOLATE(multiplier) ({\
    UPDATE(multiplier, 0);\
    UPDATE(multiplier, 1);\
    UPDATE(multiplier, 2);\
})
#elif NUM_MATERIALS == 4
#define INTERPOLATE(multiplier) ({\
    UPDATE(multiplier, 0);\
    UPDATE(multiplier, 1);\
    UPDATE(multiplier, 2);\
    UPDATE(multiplier, 3);\
})
#elif NUM_MATERIALS == 5
#define INTERPOLATE(multiplier) ({\
    UPDATE(multiplier, 0);\
    UPDATE(multiplier, 1);\
    UPDATE(multiplier, 2);\
    UPDATE(multiplier, 3);\
    UPDATE(multiplier, 4);\
})
#elif NUM_MATERIALS == 6
#define INTERPOLATE(multiplier) ({\
    UPDATE(multiplier, 0);\
    UPDATE(multiplier, 1);\
    UPDATE(multiplier, 2);\
    UPDATE(multiplier, 4);\
    UPDATE(multiplier, 5);\
})  
#elif NUM_MATERIALS == 7
#define INTERPOLATE(multiplier) ({\
    UPDATE(multiplier, 0);\
    UPDATE(multiplier, 1);\
    UPDATE(multiplier, 2);\
    UPDATE(multiplier, 4);\
    UPDATE(multiplier, 5);\
    UPDATE(multiplier, 6);\
})
#elif NUM_MATERIALS == 8
#define INTERPOLATE(multiplier) ({\
    UPDATE(multiplier, 0);\
    UPDATE(multiplier, 1);\
    UPDATE(multiplier, 2);\
    UPDATE(multiplier, 4);\
    UPDATE(multiplier, 5);\
    UPDATE(multiplier, 6);\
    UPDATE(multiplier, 7);\
})
#elif NUM_MATERIALS == 9
#define INTERPOLATE(multiplier) ({\
    UPDATE(multiplier, 0);\
    UPDATE(multiplier, 1);\
    UPDATE(multiplier, 2);\
    UPDATE(multiplier, 4);\
    UPDATE(multiplier, 5);\
    UPDATE(multiplier, 6);\
    UPDATE(multiplier, 7);\
    UPDATE(multiplier, 8);\
})
#elif NUM_MATERIALS == 10
#define INTERPOLATE(multiplier) ({\
    UPDATE(multiplier, 0);\
    UPDATE(multiplier, 1);\
    UPDATE(multiplier, 2);\
    UPDATE(multiplier, 4);\
    UPDATE(multiplier, 5);\
    UPDATE(multiplier, 6);\
    UPDATE(multiplier, 7);\
    UPDATE(multiplier, 8);\
    UPDATE(multiplier, 9);\
})
#elif NUM_MATERIALS == 11
#define INTERPOLATE(multiplier) ({\
    UPDATE(multiplier, 0);\
    UPDATE(multiplier, 1);\
    UPDATE(multiplier, 2);\
    UPDATE(multiplier, 4);\
    UPDATE(multiplier, 5);\
    UPDATE(multiplier, 6);\
    UPDATE(multiplier, 7);\
    UPDATE(multiplier, 8);\
    UPDATE(multiplier, 9);\
    UPDATE(multiplierl, 10);\
})
#elif NUM_MATERIALS == 12
#define INTERPOLATE(multiplier) ({\
    UPDATE(multiplier, 0);\
    UPDATE(multiplier, 1);\
    UPDATE(multiplier, 2);\
    UPDATE(multiplier, 4);\
    UPDATE(multiplier, 5);\
    UPDATE(multiplier, 6);\
    UPDATE(multiplier, 7);\
    UPDATE(multiplier, 8);\
    UPDATE(multiplier, 9);\
    UPDATE(multiplier, 10);\
    UPDATE(multiplier, 11);\
})
#elif NUM_MATERIALS == 13
#define INTERPOLATE(multiplier) ({\
    UPDATE(multiplier, 0);\
    UPDATE(multiplier, 1);\
    UPDATE(multiplier, 2);\
    UPDATE(multiplier, 4);\
    UPDATE(multiplier, 5);\
    UPDATE(multiplier, 6);\
    UPDATE(multiplier, 7);\
    UPDATE(multiplier, 8);\
    UPDATE(multiplier, 9);\
    UPDATE(multiplier, 10);\
    UPDATE(multiplier, 11);\
    UPDATE(multiplier, 12);\
})
#elif NUM_MATERIALS == 14
#define INTERPOLATE(multiplier) ({\
    UPDATE(multiplier, 0);\
    UPDATE(multiplier, 1);\
    UPDATE(multiplier, 2);\
    UPDATE(multiplier, 4);\
    UPDATE(multiplier, 5);\
    UPDATE(multiplier, 6);\
    UPDATE(multiplier, 7);\
    UPDATE(multiplier, 8);\
    UPDATE(multiplier, 9);\
    UPDATE(multiplier, 10);\
    UPDATE(multiplier, 11);\
    UPDATE(multiplier, 12);\
    UPDATE(multiplier, 13);\
})
#else
#define INTERPOLATE(multiplier) {\
    fprintf(stderr, "NUM_MATERIALS not in [1, 14]");\
)
#endif

// the CT volume (used to be tex_density)
texture<float, 3, hipReadModeElementType> volume;

extern "C" {
    __global__  void projectKernel(
        int out_width, // width of the output image
        int out_height, // height of the output image
        float step,
        float gVolumeEdgeMinPointX,
        float gVolumeEdgeMinPointY,
        float gVolumeEdgeMinPointZ,
        float gVolumeEdgeMaxPointX,
        float gVolumeEdgeMaxPointY,
        float gVolumeEdgeMaxPointZ,
        float gVoxelElementSizeX,
        float gVoxelElementSizeY,
        float gVoxelElementSizeZ,
        float sx, // x-coordinate of source point for rays in world-space
        float sy,
        float sz,
        float* rt_kinv, // (3, 3) array giving the image-to-world-ray transform.
        float* output, // flat array, with shape (out_height, out_width, NUM_MATERIALS).
        int offsetW,
        int offsetH)
    {

        // The output image has the following coordinate system, with cell-centered sampling.
        // y is along the fast axis (columns), x along the slow (rows).
        // Each point has NUM_MATERIALS elements at it.
        // 
        //      x -->
        //    y *---------------------------*
        //    | |                           |
        //    V |                           |
        //      |        output image       |
        //      |                           |
        //      |                           |
        //      *---------------------------*
        // 
        //
        int udx = threadIdx.x + (blockIdx.x + offsetW) * blockDim.x; // index into output image width
        int vdx = threadIdx.y + (blockIdx.y + offsetH) * blockDim.y; // index into output image height

        // if the current point is outside the output image, no computation needed
        if (udx >= out_width || vdx >= out_height)
            return;

        // flat index to first material in output "channel". 
        // So (idx + m) gets you the pixel for material index m in [0, NUM_MATERIALS)
        int idx = udx * (out_height * NUM_MATERIALS) + vdx * NUM_MATERIALS; 

        // cell-centered sampling point corresponding to pixel index, in index-space.
        float u = (float) udx + 0.5;
        float v = (float) vdx + 0.5;

        // Vector in voxel-space along ray from source-point to pixel at [u,v] on the detector plane.
        float rx = u * rt_kinv[0] + v * rt_kinv[1] + rt_kinv[2];
        float ry = u * rt_kinv[3] + v * rt_kinv[4] + rt_kinv[5];
        float rz = u * rt_kinv[6] + v * rt_kinv[7] + rt_kinv[8];

        // make the ray a unit vector
        float normFactor = 1.0f / (sqrt((rx * rx) + (ry * ry) + (rz * rz)));
        rx *= normFactor;
        ry *= normFactor;
        rz *= normFactor;

        // calculate projections
        // Part 1: compute alpha value at entry and exit point of the volume on either side of the ray.
        // minAlpha: the distance from source point to volume entry point of the ray.
        // maxAlpha: the distance from source point to volume exit point of the ray.
        float minAlpha, maxAlpha;
        minAlpha = 0;
        maxAlpha = INFINITY;

        if (0.0f != rx)
        {
            float reci = 1.0f / rx;
            float alpha0 = (gVolumeEdgeMinPointX - sx) * reci;
            float alpha1 = (gVolumeEdgeMaxPointX - sx) * reci;
            minAlpha = fmin(alpha0, alpha1);
            maxAlpha = fmax(alpha0, alpha1);
        }
        else if (gVolumeEdgeMinPointX > sx || sx > gVolumeEdgeMaxPointX)
        {
            return;
        }

        if (0.0f != ry)
        {
            float reci = 1.0f / ry;
            float alpha0 = (gVolumeEdgeMinPointY - sy) * reci;
            float alpha1 = (gVolumeEdgeMaxPointY - sy) * reci;
            minAlpha = fmax(minAlpha, fmin(alpha0, alpha1));
            maxAlpha = fmin(maxAlpha, fmax(alpha0, alpha1));
        }
        else if (gVolumeEdgeMinPointY > sy || sy > gVolumeEdgeMaxPointY)
        {
            return;
        }

        if (0.0f != rz)
        {
            float reci = 1.0f / rz;
            float alpha0 = (gVolumeEdgeMinPointZ - sz) * reci;
            float alpha1 = (gVolumeEdgeMaxPointZ - sz) * reci;
            minAlpha = fmax(minAlpha, fmin(alpha0, alpha1));
            maxAlpha = fmin(maxAlpha, fmax(alpha0, alpha1));
        }
        else if (gVolumeEdgeMinPointZ > sz || sz > gVolumeEdgeMaxPointZ)
        {
            return;
        }

        // we start not at the exact entry point 
        // => we can be sure to be inside the volume
        // (this is commented out intentionally, seemingly)
        //minAlpha += step * 0.5f;
        
        // Part 2: Cast ray if it intersects the volume

        // Trapezoidal rule (interpolating function = piecewise linear func)
        float px, py, pz; // voxel-space point
        int t; // number of steps along ray
        float alpha; // distance along ray (alpha = minAlpha + step * t)
        float boundary_factor; // factor to multiply at the boundary.

        // initialize the output to 0.
        for (int m = 0; m < NUM_MATERIALS; m++) {
            output[idx + m] = 0;
        }

        // Sample the points along the ray at the entrance boundary of the volume and the mid segments.
        for (t = 0, alpha = minAlpha; alpha < maxAlpha; t++, alpha += step)
        {
            // Get the current sample point in the volume voxel-space.
            // In CUDA, voxel centeras are located at (xx.5, xx.5, xx.5), whereas SwVolume has voxel centers at integers.
            px = sx + alpha * rx + 0.5;
            py = sy + alpha * ry + 0.5;
            pz = sz + alpha * rz - gVolumeEdgeMinPointZ;

            /* For the entry boundary, multiply by 0.5 (this is the t == 0 check). That is, for the initial interpolated value, 
             * only a half step-size is considered in the computation.
             * For the second-to-last interpolation point, also multiply by 0.5, since there will be a final step at the maxAlpha boundary.
             */ 
            boundary_factor = (t == 0 || alpha + step >= maxAlpha) ? 0.5 : 1.0;

            // Perform the interpolation. This involves the variables: output, idx, px, py, pz, and volume. 
            // It is done for each segmentation.
            INTERPOLATE(boundary_factor);
        }

        // Scaling by step;
        output[idx] *= step;

        // Last segment of the line
        if (output[idx] > 0.0f) {
            alpha -= step;
            float lastStepsize = maxAlpha - alpha;

            // scaled last step interpolation (something weird?)
            INTERPOLATE(0.5 * lastStepsize);

            // The last segment of the line integral takes care of the varying length.
            px = sx + alpha * rx + 0.5;
            py = sy + alpha * ry + 0.5;
            pz = sz + alpha * rz - gVolumeEdgeMinPointZ;

            // interpolation
            INTERPOLATE(0.5 * lastStepsize);
        }

        // normalize output value to world coordinate system units
        for (int m = 0; m < NUM_MATERIALS; m++) {
            output[idx + m] *= sqrt((rx * gVoxelElementSizeX)*(rx * gVoxelElementSizeX) + (ry * gVoxelElementSizeY)*(ry * gVoxelElementSizeY) + (rz * gVoxelElementSizeZ)*(rz * gVoxelElementSizeZ));
        }
    
        return;
    }
}
    
